#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_drvapi.h>
#include <helper_functions.h>
#include <helper_image.h>
#include <hip/hip_vector_types.h>
#include <helper_string.h>
#include <helper_timer.h>

#define PI 3.14159265358979323846
#define STR_MAX_LEN 256
#define frand() (rand()/(RAND_MAX+1.0))

void setting(int *l, float *rx, float *ry, float *rz, int n)
{
  int i, rem, nbs, r, *bas; 

  *l = (int)sqrt((float)n);
  if((*l)*(*l) < n) *l += 1;

  srand(time(NULL));

  nbs = (*l)*(*l);
  bas = (int *)malloc(nbs*sizeof(int));
  for(i = 0; i < nbs; i++) bas[i] = i;
  rem = nbs - n;
  while(rem > 0){
    r = (int)(nbs*frand());
    for(i = r; i < nbs-1; i++) bas[i] = bas[i+1];
    nbs--;
    rem--;
  }

  for(i = 0; i < n; i++){
    rx[i] = bas[i]%(*l);
    ry[i] = bas[i]/(*l);
    rz[i] = (*l)/2;
    rx[i] += 0.05*sqrt(-2.0*log(frand()))*cos(2.0*M_PI*frand()); //Box Muller Transform
    ry[i] += 0.05*sqrt(-2.0*log(frand()))*cos(2.0*M_PI*frand());
    rz[i] += 0.05*sqrt(-2.0*log(frand()))*cos(2.0*M_PI*frand());
    if(rx[i] < 0.0) rx[i] += *l;
    if(ry[i] < 0.0) ry[i] += *l;
    if(rz[i] < 0.0) rz[i] += *l;
    if(rx[i] >  *l) rx[i] -= *l;
    if(ry[i] >  *l) ry[i] -= *l;
    if(rz[i] >  *l) rz[i] -= *l;
  }

  free(bas);
}

void outdata(FILE *fp, int n, int l,
  float *rx, float *ry, float *rz, float *vx, float *vy, float *vz, int step)
{
  int i, j;
  float rc;
  float dx, dy, dz, dr;  
  float et, ek, ev;  

  rc = l/2.0;
  ek = 0.0;
  ev = 0.0;
  for(i = 0; i < n; i++){
    ek += (vx[i]*vx[i] + vy[i]*vy[i] + vz[i]*vz[i])/2.0; 
    for(j = i+1; j < n; j++){
      dx = rx[i] - rx[j];
      dy = ry[i] - ry[j];
      dz = rz[i] - rz[j];
      if(dx < -rc) dx += l;
      if(dy < -rc) dy += l;
      if(dz < -rc) dz += l;
      if(dx >  rc) dx -= l;
      if(dy >  rc) dy -= l;
      if(dz >  rc) dz -= l;
      dr = sqrt(dx*dx + dy*dy + dz*dz);
      if(dr < rc) ev += (1.0/pow(dr,12) - 2.0/pow(dr,6))/12.0; //Lennard-Jones
    }
  }
  et = ek + ev;

  fprintf(fp, "%i %f %f %f\n", step, ek, ev, et);
}

void outaxsf(FILE *gp, int n, int l, float dt,
  float *rx, float *ry, float *rz, float *fx, float *fy, float *fz, int nstep, int step)
{
  int i;

  if(step == 0){
    fprintf(gp, "ANIMSTEPS %i\n", nstep);
    fprintf(gp, "SLAB\n");
    fprintf(gp, "PRIMVEC\n");
    fprintf(gp, "%f 0.0 0.0\n", (float)l);
    fprintf(gp, "0.0 %f 0.0\n", (float)l);
    fprintf(gp, "0.0 0.0 1.0\n");
  }
  fprintf(gp, "PRIMCOORD %i\n", step+1);
  fprintf(gp, "%i 1\n", n);

  for(i = 0; i < n; i++){
    fprintf(gp, "%g %g %g\n", rx[i], ry[i], rz[i]);
  }

  fflush(gp);
}

__global__ void force(int thr, int n, int l,
  float *fx, float *fy, float *fz, float *rx, float *ry, float *rz)
{
  int i, j;
  float dx, dy, dz, dr;
  float rc, fr;
#ifdef _SHR
  int k, blk;
  __shared__ float shrx[64], shry[64], shrz[64];
#endif

  if((i = thr*blockIdx.x * blockDim.x + threadIdx.x) > n) return;

  rc = l/2.0;
  fx[i] = 0.0;
  fy[i] = 0.0;
  fz[i] = 0.0;
#ifdef _SHR
  blk = n/thr + (n%thr ? 1 : 0);

  for(k = 0; k < blk; k++){
    __syncthreads();   
    shrx[threadIdx.x] = rx[thr*k+threadIdx.x];
    shry[threadIdx.x] = ry[thr*k+threadIdx.x];
    shrz[threadIdx.x] = rz[thr*k+threadIdx.x];
    __syncthreads();   
    for(j = 0; j < thr; j++){
      if(i == thr*k+j) continue;
      dx = rx[i] - shrx[j];
      dy = ry[i] - shry[j];
      dz = rz[i] - shrz[j];
      if(dx < -rc) dx += l;
      if(dy < -rc) dy += l;
      if(dz < -rc) dz += l;
      if(dx >  rc) dx -= l;
      if(dy >  rc) dy -= l;
      if(dz >  rc) dz -= l;
      dr = sqrt(dx*dx + dy*dy + dz*dz); //Resultan
      if(dr < rc){
        fr = 1.0/pow(dr,13) - 1.0/pow(dr,7); //Force: derivation of Potential En. (Lennard-Jones)
        fx[i] += fr*dx/dr;
        fy[i] += fr*dy/dr;
        fz[i] += fr*dz/dr;
      }
    }
  }
#else
  for(j = 0; j < n; j++){
    if(i == j) continue;
    dx = rx[i] - rx[j];
    dy = ry[i] - ry[j];
    dz = rz[i] - rz[j];
    if(dx < -rc) dx += l;
    if(dy < -rc) dy += l;
    if(dz < -rc) dz += l;
    if(dx >  rc) dx -= l;
    if(dy >  rc) dy -= l;
    if(dz >  rc) dz -= l;
    dr = sqrt(dx*dx + dy*dy + dz*dz);
    if(dr < rc){
      fr = 1.0/pow(dr,13) - 1.0/pow(dr,7);
      fx[i] += fr*dx/dr;
      fy[i] += fr*dy/dr;
      fz[i] += fr*dz/dr;
    }
  }
#endif
}

int main(int argc, char **argv)
{
  char str[STR_MAX_LEN];
  int i, step, size;
  int l, na, nstep, blk, thr;
  float dt;
  float *rx, *ry, *rz, *vx, *vy, *vz, *fx, *fy, *fz, *ax, *ay, *az;
  float *dev_rx, *dev_ry, *dev_rz, *dev_fx, *dev_fy, *dev_fz;
  float m = 100;
  FILE *fp,*ip, *op, *gp, *input;

  input=fopen("md.in", "r");
  for(i = 0; fgets(str, STR_MAX_LEN, input) != NULL; i++){
    if(*str == '#') fgets(str, STR_MAX_LEN, input);
    switch(i){
      case 0:
        sscanf(str, "%i", &na);
        break;
      case 1:
        sscanf(str, "%i", &nstep);
        break;
      case 2:
        sscanf(str, "%g", &dt);
        break;
      default:
        printf("error in md.in\n");
        return 0;
    }
  }
  
  fclose(input);

  if(argc == 2){
    thr = atoi(argv[1]);
    if(thr > na) thr = na;
    blk = na/thr + (na%thr ? 1 : 0);
  } else{
    printf("error\n");
    return 1;
  }

 fp=fopen("md_cuda_leap_energy.dat", "w");
 gp=fopen("md_cuda_leap_data.dat", "w");
 ip=fopen("init_pos_cuda_leap.dat", "w");
 op=fopen("final_pos_leap_cuda.dat", "w");

  size = na*sizeof(float);

  rx=(float *)malloc(size);
  ry=(float *)malloc(size);
  rz=(float *)malloc(size);
  vx=(float *)malloc(size);
  vy=(float *)malloc(size);
  vz=(float *)malloc(size);
  fx=(float *)malloc(size);
  fy=(float *)malloc(size);
  fz=(float *)malloc(size);
  ax=(float *)malloc(size);
  ay=(float *)malloc(size);
  az=(float *)malloc(size);

  setting(&l, rx, ry, rz, na);
  
  for(i = 0; i < na; i++){
  fprintf(ip, "%f %f %f\n", rx[i], ry[i], rz[i]);
  } 

  for(i = 0; i < na; i++){
    vx[i] = 0.0;
    vy[i] = 0.0;
    vz[i] = 0.0;
  }

  hipMalloc((void **)&dev_rx, size);
  hipMalloc((void **)&dev_ry, size);
  hipMalloc((void **)&dev_rz, size);
  hipMalloc((void **)&dev_fx, size);
  hipMalloc((void **)&dev_fy, size);
  hipMalloc((void **)&dev_fz, size);

  hipMemcpy(dev_rx, rx, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_ry, ry, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_rz, rz, size, hipMemcpyHostToDevice);

  force<<<blk,thr>>>(thr, na, l, dev_fx, dev_fy, dev_fz, dev_rx, dev_ry, dev_rz);

  hipDeviceSynchronize();

  hipMemcpy(fx, dev_fx, size, hipMemcpyDeviceToHost);
  hipMemcpy(fy, dev_fy, size, hipMemcpyDeviceToHost);
  hipMemcpy(fz, dev_fz, size, hipMemcpyDeviceToHost);

  outdata(fp, na, l, rx, ry, rz, vx, vy, vz, 0);

  for(step = 1; step < nstep; step++){
    printf("step = %d\r",step);    
    fflush(stdout);
    for(i = 0; i < na; i++){
        ax[i] = fx[i] / m;
        ay[i] = fy[i] / m;
        az[i] = fz[i] / m;

        vx[i] += 0.5 * ax[i] * dt;
        vy[i] += 0.5 * ay[i] * dt;
        vz[i] += 0.5 * az[i] * dt;
    }

    for(i = 0; i < na; i++){
     
        rx[i] += vx[i] * dt;
        ry[i] += vy[i] * dt;
        rz[i] += vz[i] * dt;

        if(rx[i] < 0.0) rx[i] += l;
        if(ry[i] < 0.0) ry[i] += l;
        if(rz[i] < 0.0) rz[i] += l;
        if(rx[i] >   l) rx[i] -= l;
        if(ry[i] >   l) ry[i] -= l;
        if(rz[i] >   l) rz[i] -= l;
    }
    hipMemcpy(dev_rx, rx, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_ry, ry, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_rz, rz, size, hipMemcpyHostToDevice);

    force<<<blk,thr>>>(thr, na, l, dev_fx, dev_fy, dev_fz, dev_rx, dev_ry, dev_rz);

    hipDeviceSynchronize();

    hipMemcpy(fx, dev_fx, size, hipMemcpyDeviceToHost);
    hipMemcpy(fy, dev_fy, size, hipMemcpyDeviceToHost);
    hipMemcpy(fz, dev_fz, size, hipMemcpyDeviceToHost);
    
    for(i = 0; i < na; i++){

      ax[i] = fx[i] / m;
      ay[i] = fy[i] / m;
      az[i] = fz[i] / m;

      vx[i] += (0.5 * ax[i] * dt);
      vy[i] += (0.5 * ay[i] * dt);
      vz[i] += (0.5 * az[i] * dt);
    }

    outdata(fp, na, l, rx, ry, rz, vx, vy, vz, step);
    outaxsf(gp, na, l, dt, rx, ry, rz, fx, fy, fz, nstep, nstep);
  }
  for(i = 0; i < na; i++){
  fprintf(op, "%f %f %f\n", rx[i], ry[i], rz[i]);
  }

  hipFree(&dev_rx);
  hipFree(&dev_ry);
  hipFree(&dev_rz);
  hipFree(&dev_fx);
  hipFree(&dev_fy);
  hipFree(&dev_fz);

  free(rx);
  free(ry);
  free(rz);
  free(vx);
  free(vy);
  free(vz);
  free(fx);
  free(fy);
  free(fz);

  fclose(fp);
 
  return 0;
}


